#include "hip/hip_runtime.h"
// input: radius (1), nsample (1), xyz1 (b,n,3), xyz2 (b,m,3)
// output: idx (b,m,nsample), pts_cnt (b,m)
__global__ void query_ball_point_gpu(int b, int n, int m, float radius, int nsample, const float *xyz1, const float *xyz2, int *idx, int *pts_cnt) {
    int batch_index = blockIdx.x;
    xyz1 += n*3*batch_index;
    xyz2 += m*3*batch_index;
    idx += m*nsample*batch_index;
    pts_cnt += m*batch_index; // counting how many unique points selected in local region

    int index = threadIdx.x;
    int stride = blockDim.x;
    
    for (int j=index;j<m;j+=stride) {
        int cnt = 0;
        for (int k=0;k<n;++k) {
            if (cnt == nsample)
                break; // only pick the FIRST nsample points in the ball
            float x2=xyz2[j*3+0];
            float y2=xyz2[j*3+1];
            float z2=xyz2[j*3+2];
            float x1=xyz1[k*3+0];
            float y1=xyz1[k*3+1];
            float z1=xyz1[k*3+2];
    	    float d=max(sqrtf((x2-x1)*(x2-x1)+(y2-y1)*(y2-y1)+(z2-z1)*(z2-z1)),1e-20f);
            if (d<radius) {
                if (cnt==0) { // set ALL indices to k, s.t. if there are less points in ball than nsample, we still have valid (repeating) indices
                    for (int l=0;l<nsample;++l)
                        idx[j*nsample+l] = k;
                }
                idx[j*nsample+cnt] = k;
                cnt+=1;
            }
        }
        pts_cnt[j] = cnt;
    }
}

// input: points (b,n,c), idx (b,m,nsample)
// output: out (b,m,nsample,c)
__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
    int batch_index = blockIdx.x;
    points += n*c*batch_index;
    idx += m*nsample*batch_index;
    out += m*nsample*c*batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;
    
    for (int j=index;j<m;j+=stride) {
        for (int k=0;k<nsample;++k) {
            int ii = idx[j*nsample+k];
            for (int l=0;l<c;++l) {
                out[j*nsample*c+k*c+l] = points[ii*c+l];
            }
        }
    }
}

// input: grad_out (b,m,nsample,c), idx (b,m,nsample), 
// output: grad_points (b,n,c)
__global__ void group_point_grad_gpu(int b, int n, int c, int m, int nsample, const float *grad_out, const int *idx, float *grad_points) {
    int batch_index = blockIdx.x;
    idx += m*nsample*batch_index;
    grad_out += m*nsample*c*batch_index;
    grad_points += n*c*batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    for (int j=index;j<m;j+=stride) {
        for (int k=0;k<nsample;++k) {
            int ii = idx[j*nsample+k];
            for (int l=0;l<c;++l) {
                 atomicAdd(&grad_points[ii*c+l], grad_out[j*nsample*c+k*c+l]);
            }
        }
    }
}

// input: k (1), distance matrix dist (b,m,n)
// output: idx (b,m,n), dist_out (b,m,n)
// only the top k results within n are useful

/**
 * @brief KNN函数实现的排序GPU支持

 * @in
 * @param b:in BatchSize
 * @param n:in 输入待定点的数量input points
 * @param m:IN query points 的数量
 * @param k:in KNN的k值
 * @param dist:in 输入的点位置欧式距离平法信息【Batch】[m][n]
 * @param outi:out 前k的距离点位的索引
 * @param out：out 前k的距离点位的距离
 * @return __global__ 
 */
__global__ void selection_sort_gpu(int b, int n, int m, int k, const float *dist, int *outi, float *out) {
    int batch_index = blockIdx.x;
    dist+=m*n*batch_index;//定位开始位置
    outi+=m*n*batch_index;
    out+=m*n*batch_index;


    int index = threadIdx.x; //线程id
    int stride = blockDim.x; //block大小

    // copy from dist to dist_out
    for (int j=index;j<m;j+=stride) {
        for (int s=0;s<n;++s) {
            out[j*n+s] = dist[j*n+s];
            outi[j*n+s] = s;
        }
    }

    float *p_dist;
    for (int j=index;j<m;j+=stride) {
        p_dist = out+j*n;
        // selection sort for the first k elements
        for (int s=0;s<k;++s) {
            int min=s; 
            // find the min
            for (int t=s+1;t<n;++t) {
                if (p_dist[t]<p_dist[min]) {
                    min = t;
                }
            }
            // swap min-th and i-th element
            if (min!=s) {
                float tmp = p_dist[min];
                p_dist[min] = p_dist[s];
                p_dist[s] = tmp;
                int tmpi = outi[j*n+min];
                outi[j*n+min] = outi[j*n+s];
                outi[j*n+s] = tmpi;
            }
        }
    }
}

/**
 * @brief KNN的直接gpu实现
 * 
 * @param b 
 * @param n 
 * @param m 
 * @param k 
 * @param xyz1 [b][n][3]
 * @param xyz2 [b][m][3]
 * @param outi 
 * @param out 
 * @return __global__ 
 */
__global__ void knn_kernal_gpu(int b,int n,int m,int k,const float * xyz1,const float * xyz2,float * outi,float *out){
    //TODO:实现knn核算子
    int batch_index = blockIdx.x;
    xyz1+=3*n*batch_index;
    xyz2+=3*m*batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    __shared__ int point_index[m][n];
    __shared__ float point_val[m][n]; 
    // copy from dist to dist_out
    for (int j=index;j<m;j+=stride) {
        for (int s=0;s<n;++s) {
            for(int pos =0;pos<3;pos++){
                point_val[j][s] += (xyz1[s][pos] - xyz2[j][pos])*(xyz1[s][pos] - xyz2[j][pos]);
                out[j*n+s] += (xyz1[s][pos] - xyz2[j][pos])*(xyz1[s][pos] - xyz2[j][pos]) ;//取出的是此batch中xyz1中第s个和xyz2中第j个个的距离
            }
            outi[j*n+s] = s;
            point_index[j][s]=s;
        }
    }
        //此处不需要_syncthreads()因为都是在一个block中的数据 不会互相影响
    float *p_dist;
    for (int j=index;j<m;j+=stride) {
        p_dist = out+j*n;
        // selection sort for the first k elements
        for (int s=0;s<k;++s) {
            int min=s; 
            // find the min
            for (int t=s+1;t<n;++t) {
                if (point_val[j][t]<point_val[j][min]) {
                    min = t;
                }
            }
            // swap min-th and i-th element
            if (min!=s) {
                float tmp = point_val[j][min];
                point_val[j][min] = point_val[j][s];
                point_val[j][s] = tmp;
                int tmpi = point_index[j][min];
                point_index[j][min] = point_index[j][s];
                point_index[j][s] = tmpi;
            }
            //最后将结果写入输出变量
            p_dist[s] = point_val[j][s];
            outi[j*n+s] = point_index[j][min];
        }
    }


}
void queryBallPointLauncher(int b, int n, int m, float radius, int nsample, const float *xyz1, const float *xyz2, int *idx, int *pts_cnt) {
    query_ball_point_gpu<<<b,256>>>(b,n,m,radius,nsample,xyz1,xyz2,idx,pts_cnt);
    //hipDeviceSynchronize();
}
void selectionSortLauncher(int b, int n, int m, int k, const float *dist, int *outi, float *out) {
    selection_sort_gpu<<<b,256>>>(b,n,m,k,dist,outi,out); 
    //hipDeviceSynchronize();
}
void groupPointLauncher(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out){
    group_point_gpu<<<b,256>>>(b,n,c,m,nsample,points,idx,out);
    //hipDeviceSynchronize();
}
void groupPointGradLauncher(int b, int n, int c, int m, int nsample, const float *grad_out, const int *idx, float *grad_points){
    group_point_grad_gpu<<<b,256>>>(b,n,c,m,nsample,grad_out,idx,grad_points);
    //group_point_grad_gpu<<<1,1>>>(b,n,c,m,nsample,grad_out,idx,grad_points);
    //hipDeviceSynchronize();
}

void knn_gpu(int b,int n,int m,int k,const float * xyz1,const float * xyz2,float * outi,float *out)
{
    knn_kernal_gpu<<<b,256>>>(b,n,m,k,xyz1,xyz2,outi,out);
}
